#include<iostream>
#include<vector>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;
using namespace std::chrono;


__global__ void addKernel(const int *A, const int *B, int *C, int n){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid < n)
    {
        C[tid] = A[tid] + B[tid];
    }
    
}

void vec_add(int n){
    // int id = cudaGetDevice(&id);
    int *A, *B, *C;

    hipMallocManaged(&A, n * sizeof(int));
    hipMallocManaged(&B, n * sizeof(int));
    hipMallocManaged(&C, n * sizeof(int));

    for (int i = 0; i < n; i++) {
        A[i] = i;
        B[i] = i;
    }

    auto start_clock = high_resolution_clock::now();
    for (int i = 0; i < n; i++) {
        C[i] = A[i] + B[i];
    }
    auto end_clock = high_resolution_clock::now();
    auto duration_cpu = duration_cast<milliseconds>(end_clock - start_clock);
    
    int threads = 256;
    int grid = (n + 256 - 1) / 256;

    // cudaMemPrefetchAsync(A, n * sizeof(int), id);
    // cudaMemPrefetchAsync(B, n * sizeof(int), id);

    hipEvent_t cuda_start, cuda_end;
    hipEventCreate(&cuda_start);
    hipEventCreate(&cuda_end);

    hipEventRecord(cuda_start);
    addKernel<<<grid, threads>>>(A, B, C, n);
    hipEventRecord(cuda_end);
    hipEventSynchronize(cuda_end);

    // cudaMemPrefetchAsync(B, n * sizeof(int), cudaCpuDeviceId);

    float duration_gpu = 0;
    hipEventElapsedTime(&duration_gpu, cuda_start, cuda_end);
    
    cout << "Cpu time: " << duration_cpu.count() << "Ms" << endl;
    cout << "GPU time: " << (duration_gpu) << "Ms" << endl;


    hipFree(A);
    hipFree(B);
    hipFree(C);
    
}   


int main(){

    vector<int> a ={100000, 1000000, 10000000, 100000000};

    for(int i=0; i<a.size(); i++){
        cout << "Vector size: " << a[i] << endl;
        vec_add(a[i]);
        cout << endl;
    }
    return 0;
}